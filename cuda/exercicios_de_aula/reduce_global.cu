
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define THREADS 20 // 1000 threads per block
#define ARRAYSIZE 20

__global__ void reduceSum(int *data){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    for (int stride = blockDim.x/2; stride > 0; stride >>=1) {
        if (tid < stride){
            data[index] += data[index+stride];
        }
    __syncthreads();
    }
    if(tid == 0){
        data[tid] = data[0];
    }
}

int main(){
    int *a, *d_a, lastElement=0;
    int new_array_size = ARRAYSIZE;

    a = (int*) malloc(sizeof(int) * ARRAYSIZE);    

    for(int i = 0; i < ARRAYSIZE; i++){
        a[i] = i;
    }
    printf("\nEntrada\n");
    for(int i = 0; i < ARRAYSIZE; i++){
        printf("%d ", a[i]);
    }
    if (ARRAYSIZE % 2 != 0){
        new_array_size--;
        lastElement = a[new_array_size];
        a = (int*) realloc(a, sizeof(int) * new_array_size);  
    }

    printf("\nEntrada normalizada\n");
    
    for(int i = 0; i < new_array_size; i++){
        printf("%d ", a[i]);
    }       

    hipMalloc((void**) &d_a, sizeof(int) * new_array_size);
    hipMemcpy(d_a, a, sizeof(int) * new_array_size, hipMemcpyHostToDevice);
    int numberOfBlocks = ceil((float) new_array_size/THREADS);
    printf("\nblocks: %d\n", numberOfBlocks);
    reduceSum<<<numberOfBlocks,THREADS>>>(d_a);

    hipMemcpy(a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    printf("\nSaida: \n");
    a[0] = a[0] + lastElement;
    printf("soma = %d ", a[0]);
    
    free(a);
    hipFree(d_a);
    printf("\nFIM");

    return 0;
}